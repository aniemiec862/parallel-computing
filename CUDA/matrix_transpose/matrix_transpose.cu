
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include <string>

struct GpuTimer
{
      hipEvent_t start;
      hipEvent_t stop;

      GpuTimer()
      {
            hipEventCreate(&start);
            hipEventCreate(&stop);
      }

      ~GpuTimer()
      {
            hipEventDestroy(start);
            hipEventDestroy(stop);
      }

      void Start()
      {
            hipEventRecord(start, 0);
      }

      void Stop()
      {
            hipEventRecord(stop, 0);
      }

      float Elapsed()
      {
            float elapsed;
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
            return elapsed;
      }
};

__global__ void matrix_transpose_naive(int *input, int *output, int grid_size) {

	int indexX = threadIdx.x + blockIdx.x * blockDim.x;
	int indexY = threadIdx.y + blockIdx.y * blockDim.y;
	int index = indexY * grid_size + indexX;
	int transposedIndex = indexX * grid_size + indexY;

	output[transposedIndex] = input[index];

	// output[index] = input[transposedIndex];
}

__global__ void matrix_transpose_shared(int *input, int *output, int grid_size) {
    int block_size = blockDim.x;

    extern __shared__ int sharedMemory[];

    int indexX = threadIdx.x + blockIdx.x * blockDim.x;
    int indexY = threadIdx.y + blockIdx.y * blockDim.y;

    int tindexX = threadIdx.x + blockIdx.y * blockDim.x;
    int tindexY = threadIdx.y + blockIdx.x * blockDim.y;

    int localIndexX = threadIdx.x;
    int localIndexY = threadIdx.y;

    int index = indexY * grid_size + indexX;
    int transposedIndex = tindexY * grid_size + tindexX;

    if (threadIdx.x < block_size && threadIdx.y < block_size) {
        sharedMemory[localIndexX * block_size + localIndexY] = input[index];
    }

    __syncthreads();

    output[transposedIndex] = sharedMemory[localIndexY * block_size + localIndexX];
}

void fill_array(int *data, int grid_size) {
	for(int idx=0;idx<(grid_size*grid_size);idx++)
		data[idx] = idx;
}

int matrix_transpose(int test_type, int grid_size, int block_size) {
	int *a, *b;
    int *d_a, *d_b; // device copies of a, b, c

	int size = grid_size * grid_size *sizeof(int);

	a = (int *)malloc(size); fill_array(a, grid_size);
	b = (int *)malloc(size);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	dim3 blockSize(block_size,block_size,1);
	dim3 gridSize(grid_size/block_size,grid_size/block_size,1);

	GpuTimer timer;
	timer.Start();

    if (test_type == 0) {
        matrix_transpose_naive<<<gridSize,blockSize>>>(d_a, d_b, grid_size);
    } else {
        matrix_transpose_shared<<<gridSize,blockSize>>>(d_a, d_b, grid_size);
    }

	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

	free(a);
	free(b);
    hipFree(d_a);
	hipFree(d_b);
}

int main(void) {
	int grid_sizes[5] = {1024, 2048, 4096, 8192, 16384};
	int block_sizes[5] = {64, 128, 256, 512, 1024};
	int test_types = 2;
	int number_of_retries = 5;

	std::cout << "grid_size;block_size;type;time" << std::endl;

	for (int i = 0; i < sizeof(grid_sizes) / sizeof(grid_sizes)[0]; i++) {
  		for (int j = 0; j < sizeof(block_sizes) / sizeof(block_sizes[0]); j++) {
            for (int test_type = 0; test_type < test_types; test_type++) {
                for (int l = 0; l < number_of_retries; l++) {
                    GpuTimer timer;
                    timer.Start();

                    matrix_transpose(test_type, grid_sizes[i], block_sizes[j]);
                    timer.Stop();

                    float elapsed = timer.Elapsed();
                    std::string type;
                    if (test_type == 0) {
                        type = "naive";
                    } else {
                        type = "shared";
                    }
                    std::cout << grid_sizes[i] << ";" << block_sizes[j] << ";" << type << ";" << elapsed << std::endl;
                }
            }
		}
	}
	return 0;
}