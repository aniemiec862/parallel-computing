
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<iostream>

struct GpuTimer
{
      hipEvent_t start;
      hipEvent_t stop;

      GpuTimer()
      {
            hipEventCreate(&start);
            hipEventCreate(&stop);
      }

      ~GpuTimer()
      {
            hipEventDestroy(start);
            hipEventDestroy(stop);
      }

      void Start()
      {
            hipEventRecord(start, 0);
      }

      void Stop()
      {
            hipEventRecord(stop, 0);
      }

      float Elapsed()
      {
            float elapsed;
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
            return elapsed;
      }
};

void host_add(int *a, int *b, int *c, int N) {
	for(int idx=0;idx<N;idx++)
		c[idx] = a[idx] + b[idx];
}

__global__ void device_add(int *a, int *b, int *c) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
        c[index] = a[index] + b[index];
}


void fill_array(int *data, int N) {
	for(int idx=0;idx<N;idx++)
		data[idx] = idx;
}

void print_output(int *a, int *b, int*c, int N) {
	for(int idx=0;idx<N;idx++)
		printf("\n %d + %d  = %d",  a[idx] , b[idx], c[idx]);
}

void add_vectors(int threads_per_block, int N) {
	int *a, *b, *c;
    int *d_a, *d_b, *d_c;
	int no_of_blocks=0;

	int size = N * sizeof(int);

	a = (int *)malloc(size); fill_array(a, N);
	b = (int *)malloc(size); fill_array(b, N);
	c = (int *)malloc(size);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	no_of_blocks = N/threads_per_block;

	device_add<<<no_of_blocks,threads_per_block>>>(d_a,d_b,d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	// print_output(a,b,c, N);

	free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}

int main(void) {
	int threads_per_block[7] = {1, 2, 4, 16, 64, 128, 256};
	int vector_sizes[5] = {3355443, 3355443 * 2, 3355443 * 3, 3355443 * 4, 3355443 * 5};
	int number_of_retries = 5;

    std::cout << "threads_per_block;vector_size;time" << std::endl;

	for (int i = 0; i < sizeof(threads_per_block) / sizeof(threads_per_block)[0]; i++) {
        for (int j = 0; j < sizeof(vector_sizes) / sizeof(vector_sizes[0]); j++) {
		    for (int k = 0; k < number_of_retries; k++) {
	            GpuTimer timer;
                timer.Start();

                add_vectors(threads_per_block[i], vector_sizes[j]);
                timer.Stop();

                float elapsed = timer.Elapsed();
                std::cout << threads_per_block[i] << ";" << vector_sizes[j] << ";" << elapsed << std::endl;
			}
		}
	}
	return 0;
}
