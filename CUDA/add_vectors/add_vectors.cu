#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<stdlib.h>
#include<iostream>

struct GpuTimer
{
      hipEvent_t start;
      hipEvent_t stop;

      GpuTimer()
      {
            hipEventCreate(&start);
            hipEventCreate(&stop);
      }

      ~GpuTimer()
      {
            hipEventDestroy(start);
            hipEventDestroy(stop);
      }

      void Start()
      {
            hipEventRecord(start, 0);
      }

      void Stop()
      {
            hipEventRecord(stop, 0);
      }

      float Elapsed()
      {
            float elapsed;
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
            return elapsed;
      }
};

void host_add(int *a, int *b, int *c, int N) {
	for(int idx=0;idx<N;idx++)
		c[idx] = a[idx] + b[idx];
}

__global__ void device_add(int *a, int *b, int *c) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
        c[index] = a[index] + b[index];
}


void fill_array(int *data, int N) {
	for(int idx=0;idx<N;idx++)
		data[idx] = idx;
}

void print_output(int *a, int *b, int*c, int N) {
	for(int idx=0;idx<N;idx++)
		printf("\n %d + %d  = %d",  a[idx] , b[idx], c[idx]);
}

void add_vectors(int threads_per_block, int N) {
	int *a, *b, *c;
    int *d_a, *d_b, *d_c;
	int no_of_blocks=0;

	int size = N * sizeof(int);

	a = (int *)malloc(size); fill_array(a, N);
	b = (int *)malloc(size); fill_array(b, N);
	c = (int *)malloc(size);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	GpuTimer timer;
	no_of_blocks = N/threads_per_block;

	timer.Start();

	device_add<<<no_of_blocks,threads_per_block>>>(d_a,d_b,d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	timer.Stop();

	// print_output(a,b,c, N);

	free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

	float elapsed = timer.Elapsed();
    std::cout << threads_per_block << ";" << N << ";" << elapsed << std::endl;
}

int main(void) {
	int threads_per_block[5] = {4, 16, 64, 128, 256};
	int vector_sizes[5] = {3355443, 3355443 * 2, 3355443 * 3, 3355443 * 4, 3355443 * 5};
	int number_of_retries = 5;

    std::cout << "threads_per_block;vector_size;time" << std::endl;

	for (int i = 0; i < sizeof(threads_per_block) / sizeof(threads_per_block)[0]; i++) {
			for (int j = 0; j < sizeof(vector_sizes) / sizeof(vector_sizes[0]); j++) {
					for (int k = 0; k < number_of_retries; k++) {
						add_vectors(threads_per_block[i], vector_sizes[j]);
					}
			}
	}

	return 0;
}
